/**
    cudacal.h
    Purpose: a simple CUDA example

    @author Fan Gong
    @version 1.0 07/03/18 
*/


#include <hip/hip_runtime.h>
#include <stdexcept>

/**
    device code to calculate sqr of x.
    In CUDA, device code is prefixed with "__device__", which only runs in GPU. It can only be called by other device code or kernel code.
	Sometimes, the code prefixed with both "__device__" and "__host__" can be called by device code, kernel code or host code.

    @param x The input number.
    @return The square of the input
*/
__device__ static double _cuda_sqr(double x){
  return x * x;
}

/**
    kernel code to calculate sqr of x array.
    In CUDA, kernal code is prefixed with "__global__", which can be called from host code in the form of "<<<blocksPerGrid, threadsPerBlock>>>"

    @param n The length of x and y.
    @param x The input array.
    @param y The output array.
    @param run_num The number of iterations.
    @return none
*/
__global__ void _cuda_vdSqr(int n, double *x, double *y, int run_num){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
    for (int j = 0; j < run_num; ++j) y[i] = _cuda_sqr(x[i]);
  }
}

/**
    error check function.

    @param err The return value of a CUDA function.
    @return none
*/
static void errChk(hipError_t err){
  if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
}

/**
    host code.
    In CUDA, host code is the code that runs in the CPU, which calls the kernel code to do GPU calculation.
	It can be prefixed by "__host__" which is unnecessary since it is the default.

    @param n The length of x and y.
    @param x The input array.
    @param y The output array.
    @param run_num The number of iterations.
    @param threadsPerBlock The threads in each block. 256 is a good start.
    @param blocksPerGrid The blocks in each grid.
    @return none
*/
void cuda_vdSqr(int n, double *x, double *y, int run_num, int threadsPerBlock, int blocksPerGrid){
  size_t size = n * sizeof(double);
  double *d_x = nullptr;
  double *d_y = nullptr;
  errChk(hipMalloc((void**)&d_x, size));
  errChk(hipMalloc((void**)&d_y, size));
  
  errChk(hipMemcpy(d_x, x, size, hipMemcpyHostToDevice));
  _cuda_vdSqr<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y, run_num);
  
  errChk(hipGetLastError());
  errChk(hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost));
  
  errChk(hipFree(d_x));
  errChk(hipFree(d_y));
} 