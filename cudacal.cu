
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void _cuda_vdSqr(int n, double *x, double *y, int run_num){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
    for (int j = 0; j < run_num; ++j) y[i] = x[i] * x[i];
  }
}

static void errChk(hipError_t err){
  if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
}

void cuda_vdSqr(int n, double *x, double *y, int run_num, int threadsPerBlock, int blocksPerGrid){
  size_t size = n * sizeof(double);
  double *d_x = nullptr;
  double *d_y = nullptr;
  errChk(hipMalloc((void**)&d_x, size));
  errChk(hipMalloc((void**)&d_y, size));
  
  errChk(hipMemcpy(d_x, x, size, hipMemcpyHostToDevice));
  _cuda_vdSqr<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y, run_num);
  
  errChk(hipGetLastError());
  errChk(hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost));
  
  errChk(hipFree(d_x));
  errChk(hipFree(d_y));
} 